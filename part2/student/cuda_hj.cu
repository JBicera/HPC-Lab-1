#include "hip/hip_runtime.h"
#include <unistd.h>
#include "listutils.h"

__global__ void computeLocalRanksKernel(long* dNext, long* dRank, long* dOrderedHeadNodes, long* dSublistSizes, size_t s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= s) return;

    long current = dOrderedHeadNodes[i];
    long localRank = 0;
    long sublistSize = 0;

    while (current != -1) {
        dRank[current] = localRank;
        printf("Thread %d: Node %ld gets rank %ld\n", i, current, localRank);
        localRank++;
        sublistSize++;

        current = dNext[current];

        if (i < s - 1 && current == dOrderedHeadNodes[i + 1]) {
            break;
        }
    }

    dSublistSizes[i] = sublistSize;
}

extern "C" void parallelListRanks(const long head, const long* next, long* rank, const size_t n)
{
    // Get GPU properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Get properties of GPU 0
    size_t numSMs = prop.multiProcessorCount; // Number of SMs
    size_t s = numSMs + (numSMs / 2);

    // Device pointers
    long *dNext, *dRank, *dOrderedHeadNodes, *dSublistSizes;

    // Assume memory is allocated and initialized elsewhere
    hipMalloc((void**)&dNext, n * sizeof(long));
    hipMalloc((void**)&dRank, n * sizeof(long));

    // Copy data to device
    hipMemcpy(dNext, next, n * sizeof(long), hipMemcpyHostToDevice);
    hipMemset(dRank, 0, n * sizeof(long)); // Initialize rank array

    // Step 1: Select head nodes
    long* headNodes = (long*)malloc(s * sizeof(long));
    headNodes[0] = head;  // Ensure true head is included
    srand(42);
    int* used = (int*)calloc(n, sizeof(int));
    used[head] = 1;
    for (size_t i = 1; i < s; i++) {
        size_t idx;
        do {
            idx = rand() % n;
        } while (used[idx] == 1);
        headNodes[i] = idx;
        used[idx] = 1;
    }
    free(used);
    // Create orderedHeadNodes
    char* inHead = (char*)calloc(n, sizeof(char));
    for (size_t i = 0; i < s; i++) {
        inHead[headNodes[i]] = 1;
    }
    long* orderedHeadNodes = (long*)malloc(s * sizeof(long));
    size_t count = 0;
    long current = head;
    while (current != -1 && count < s) {
        if (inHead[current]) {
            orderedHeadNodes[count] = current;
            count++;
        }
        current = next[current];
    }
    printf("Ordered head nodes: ");
    for (size_t i = 0; i < s; i++) {
        printf("%ld ", orderedHeadNodes[i]);
    }
    printf("\n");

    free(inHead);
    free(headNodes);
    /*
    hipMalloc((void**)&dOrderedHeadNodes, s * sizeof(long));
    hipMemcpy(dOrderedHeadNodes, orderedHeadNodes, s * sizeof(long), hipMemcpyHostToDevice);

    // Allocate and copy sublistSizes
    long* sublistSizes = (long*)malloc(s * sizeof(long));
    hipMalloc((void**)&dSublistSizes, s * sizeof(long));
    hipMemset(dSublistSizes, 0, s * sizeof(long));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (s + threadsPerBlock - 1) / threadsPerBlock;
    
    // Step 2
    computeLocalRanksKernel<<<blocksPerGrid, threadsPerBlock>>>(dNext, dRank, dOrderedHeadNodes, dSublistSizes, s);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("computeLocalRanksKernel failed: %s\n", hipGetErrorString(err));
    }
    

    // Copy results back to host
    hipMemcpy(rank, dRank, n * sizeof(long), hipMemcpyDeviceToHost);
    hipMemcpy(sublistSizes, dSublistSizes, s * sizeof(long), hipMemcpyDeviceToHost);
    
    printf("Final rank array: ");
    for (size_t i = 0; i < n; i++) {
        printf("%ld ", rank[i]);
    }
    printf("\n");

    // Cleanup memory
    free(sublistSizes);
    hipFree(dNext);
    hipFree(dRank);
    hipFree(dOrderedHeadNodes);
    hipFree(dSublistSizes);
    */
}
